#include "hip/hip_runtime.h"
#include "sample.h"

__device__
Vector3 sample_hemisphere_cosine(float u1, float u2)
{
	float r = sqrtf(u1);
	float t = 2 * PI * u2;

	float x = r * cosf(t);
	float y = r * sinf(t);

	return vector3_create(x, sqrtf(1 - u1), y);
}

__device__
Vector3 sample_circle(float u1, float u2)
{
	float r = sqrtf(u1);
	float d = 2 * PI * u2;
	return vector3_create(r * cosf(d), r * sinf(d), 0);
}