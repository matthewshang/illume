#include "hip/hip_runtime.h"
#include "sphere.h"

Sphere* sphere_new(float r, Vector3 center)
{
	Sphere* sphere = (Sphere *) calloc(sizeof(Sphere), 1);
	if (!sphere)
	{
		return NULL;
	}
	sphere->r = r;
	sphere->center = center;
	return sphere;
}

Sphere sphere_create(float r, Vector3 center)
{
	Sphere sphere;
	sphere.r = r;
	sphere.center = center;
	return sphere;
}

void sphere_free(Sphere* sphere)
{
	if (sphere)
	{
		free(sphere);
	}
}

__device__
Intersection sphere_ray_intersection(Sphere* sphere, Ray* ray)
{
	 Vector3 l = vector3_sub(&sphere->center, &ray->o);
	 float s = vector3_dot(&l, &ray->d);
	 float ls = vector3_dot(&l, &l);
	 float rs = sphere->r * sphere->r;
	 if (s < 0 && ls > rs)
	 {
	 	return intersection_create_no_intersect();
	 }
	 float ms = ls - s * s;
	 if (ms > rs)
	 {
	 	return intersection_create_no_intersect();
	 }
	 float q = sqrtf(rs - ms);
	 float t = s;
	 if (ls > rs)
	 {
	 	t -= q;
	 }
	 else
	 {
	 	t += q;
	 }
	 Vector3 pos = ray_position_along(ray, t);
	 Vector3 normal = vector3_sub(&pos, &sphere->center);
	 vector3_normalize(&normal);
	 return intersection_create(1, t, normal);
}
