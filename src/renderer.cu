#include "hip/hip_runtime.h"
#include "renderer.h"

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "error_check.h"
#include "fresnel.h"
#include "material.h"
#include "microfacet.h"
#include "medium.h"
#include "primitives/sphere.h"
#include "primitives/mesh.h"
#include "primitives/mesh_instance.h"
#include "math/sample.h"
#include "math/vector3.h"
#include "math/ray.h"
#include "math/mathutils.h"
#include "math/matrix4.h"
#include "accel/bvh.h"
#include "jsonutils.h"
#include "scene/sceneref.h"

#include "intellisense.h"

#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

Renderer::Renderer(rapidjson::Value& json, HostScene& scene, int spp, int max_depth) :
    m_scene(scene), m_spp(spp), m_max_depth(max_depth)
{
	auto renderer = json.FindMember("render_settings");
	if (renderer != json.MemberEnd())
	{
		JsonUtils::from_json(renderer->value, "ray_bias", m_ray_bias);
        m_tonemapper = Tonemapper(renderer->value);
		auto res = renderer->value.FindMember("resolution");
		if (res != renderer->value.MemberEnd())
		{
			m_width = res->value.GetArray()[0].GetInt();
			m_height = res->value.GetArray()[1].GetInt();
		}
		else
		{
			printf("Renderer: resolution not found. Defaulting to (512, 512)\n");
			m_width = m_height = 512;
		}
	}
}

__global__ 
void init_curand_states(hiprandState* states, uint32_t hash, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		hiprand_init(hash + index, 0, 0, &states[index]);
	}
}

typedef struct
{
	float image_width;
	Camera camera;
	float camera_pixel_size;
	float camera_left;
	float camera_top;
} 
RenderInfo;

__global__
void init_rays(Ray* rays, int* ray_statuses, Vector3* ray_colors, Medium* ray_mediums, RenderInfo* info, hiprandState* states, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		RenderInfo i = *info;
		int x = index % (int) i.image_width;
		int y = index / (int) i.image_width;
		float left_edge = i.camera_left + i.camera_pixel_size * (float) x;
		float top_edge = i.camera_top - i.camera_pixel_size * (float) y;

		float r_x = left_edge + i.camera_pixel_size * hiprand_uniform(&states[index]);
		float r_y = top_edge - i.camera_pixel_size * hiprand_uniform(&states[index]);

		Vector3 pos = vector3_create(0, 0, 0);
		if (i.camera.aperture > FLT_EPSILON)
		{
			float u1 = hiprand_uniform(&states[index]);
			float u2 = hiprand_uniform(&states[index]);
			pos = vector3_mul(sample_circle(u1, u2), i.camera.aperture);
		}
		Vector3 origin = matrix4_mul_vector3(&i.camera.transform, pos, 1.f);
		Vector3 image_pos = matrix4_mul_vector3(&i.camera.transform, vector3_create(r_x, r_y, i.camera.dof), 1.f);
		rays[index] = ray_create(origin, vector3_sub(image_pos, origin));
		ray_statuses[index] = index;
		ray_colors[index] = vector3_create(1, 1, 1);
		ray_mediums[index] = medium_air();
	}
}

__device__
static void get_min_hit(DeviceScene* scene, Ray ray, Hit* min)
{
	min->d = FLT_MAX;
	Hit inter;
	for (int i = 0; i < scene->sphere_amount; i++)
	{
		sphere_ray_intersect(&scene->spheres[i], ray, &inter);

		if (inter.is_intersect && inter.d < min->d)
        {
			*min = inter;
		}
	}

	for (int i = 0; i < scene->instance_amount; i++)
	{
		int mesh_index = scene->instances[i].mesh_index;
		mesh_instance_ray_intersect(scene->instances + i, scene->meshes + mesh_index, ray, &inter);

		if (inter.is_intersect && inter.d < min->d)
		{
			*min = inter;
		}
	}
}

__global__
void pathtrace_kernel(Vector3* final_colors, Ray* rays, int* ray_statuses, Vector3* ray_colors, 
	Medium* ray_mediums, int depth, DeviceScene* scene, hiprandState* states, float ray_bias, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int ray_index = ray_statuses[index];

	if (index < N && ray_index != -1)
	{
		Hit min;
		hit_set_no_intersect(&min);
		get_min_hit(scene, rays[ray_index], &min);

		if (ray_mediums[ray_index].active)
		{
			Medium medium = ray_mediums[ray_index];
			float u = hiprand_uniform(&states[ray_index]);
			float scatter_dist = -log(u) / medium.scattering;
			if (scatter_dist < min.d)
			{
				float u1 = hiprand_uniform(&states[ray_index]);
				float u2 = hiprand_uniform(&states[ray_index]);
				Vector3 scatter_dir = vector3_to_basis(sample_henyey_greenstein(medium.g, u1, u2), rays[ray_index].d);
				ray_set(&rays[ray_index], ray_position_along(rays[ray_index], scatter_dist), scatter_dir);
				vector3_mul_vector_to(&ray_colors[ray_index], 
						vector3_create(expf(-1.0f * scatter_dist * medium.absorption.x),
									   expf(-1.0f * scatter_dist * medium.absorption.y),
									   expf(-1.0f * scatter_dist * medium.absorption.z)));
				if (vector3_length2(ray_colors[ray_index]) < 1e-4)
				{
					ray_statuses[index] = -1;
				}
				return;
			}
			else
			{
				vector3_mul_vector_to(&ray_colors[ray_index], 
						vector3_create(expf(-1.0f * min.d * medium.absorption.x),
									   expf(-1.0f * min.d * medium.absorption.y),
									   expf(-1.0f * min.d * medium.absorption.z)));

			}
		}

        if (min.is_intersect)
        {
            Ray r = rays[ray_index];
            Vector3 new_dir;
            Vector3 norm_o = vector3_mul(min.normal, vector3_dot(min.normal, r.d) > 0 ? -1.0f : 1.0f);
            Vector3 new_origin = ray_position_along(r, min.d);
            Vector3 albedo = min.m->albedo.eval(min.uv);

            if (min.m->type == MATERIAL_EMISSIVE)
            {
                vector3_mul_vector_to(&ray_colors[ray_index], albedo);
                vector3_add_to(&final_colors[ray_index], ray_colors[ray_index]);
                ray_statuses[index] = -1;
                new_dir = vector3_create(0, 0, 0);
            }
            else if (min.m->type == MATERIAL_DIFFUSE)
            {
                vector3_mul_vector_to(&ray_colors[ray_index], albedo);
                float u1 = hiprand_uniform(&states[ray_index]);
                float u2 = hiprand_uniform(&states[ray_index]);
                Vector3 sample = sample_hemisphere_cosine(u1, u2);
                new_dir = vector3_to_basis(sample, norm_o);
                vector3_add_to(&new_origin, vector3_mul(norm_o, ray_bias));
            }
            else if (min.m->type == MATERIAL_SPECULAR)
            {
                vector3_mul_vector_to(&ray_colors[ray_index], vector3_create(0.99f, 0.99f, 0.99f));
                new_dir = vector3_reflect(r.d, norm_o);
                vector3_add_to(&new_origin, vector3_mul(norm_o, ray_bias));
            }
            else if (min.m->type == MATERIAL_REFRACTIVE)
            {
                float cosI = -vector3_dot(r.d, min.normal);
                float cosT = 0.0f;
                float F = Fresnel::dielectric(cosI, min.m->ior, cosT);

                if (F == 1.0f || hiprand_uniform(&states[ray_index]) < F)
                {
                    new_dir = vector3_reflect(r.d, min.normal);
                    vector3_add_to(&new_origin, vector3_mul(norm_o, ray_bias));
                }
                else
                {
                    float eta = cosI < 0.0f ? min.m->ior : 1.0f / min.m->ior;
                    ray_mediums[ray_index] = cosI > 0.0f ? min.m->medium : medium_air();
                    new_dir = vector3_add(vector3_mul(r.d, eta),
                        vector3_mul(norm_o, eta * fabsf(cosI) - cosT));
                    vector3_add_to(&new_origin, vector3_mul(norm_o, -ray_bias));
                }

                vector3_mul_vector_to(&ray_colors[ray_index], albedo);
            }
            else if (min.m->type == MATERIAL_ROUGHREFLECTIVE)
            {
                Vector3 wi = vector3_mul(r.d, -1.0f);
                float wiDotN = vector3_dot(wi, min.normal);
                float a = min.m->roughness * (1.2f - 0.2f * sqrtf(fabsf(wiDotN)));

                float u1 = hiprand_uniform(&states[ray_index]);
                float u2 = hiprand_uniform(&states[ray_index]);
                Vector3 m = Microfacet::sample_Beckmann(a, u1, u2);
                m = vector3_to_basis(m, min.normal);

                float wiDotT = 0.0f;
                float wiDotM = vector3_dot(wi, m);
                float F = Fresnel::dielectric(wiDotM, min.m->ior, wiDotT);

                new_dir = vector3_reflect(r.d, m);
                if (wiDotN * vector3_dot(new_dir, min.normal) <= 0.0f)
                {
                    ray_statuses[index] = -1;
                    return;
                }
                vector3_add_to(&new_origin, vector3_mul(norm_o, ray_bias));

                float G = Microfacet::G_Beckmann(wi, new_dir, m, a);
                float weight = (F * G * fabsf(wiDotM)) / (fabsf(wiDotN) * fabsf(vector3_dot(m, min.normal)));
                vector3_mul_vector_to(&ray_colors[ray_index], vector3_mul(albedo, weight));
            }
            else if (min.m->type == MATERIAL_ROUGHREFRACTIVE)
            {
                Vector3 wi = vector3_mul(r.d, -1.0f);
                float wiDotN = vector3_dot(wi, min.normal);
                float a = min.m->roughness * (1.2f - 0.2f * sqrtf(fabsf(wiDotN)));

                float u1 = hiprand_uniform(&states[ray_index]);
                float u2 = hiprand_uniform(&states[ray_index]);
                Vector3 m = Microfacet::sample_Beckmann(a, u1, u2);
                m = vector3_to_basis(m, min.normal);

                float wiDotT = 0.0f;
                float wiDotM = vector3_dot(wi, m);
                float F = Fresnel::dielectric(wiDotM, min.m->ior, wiDotT);

                if (F == 1.0f || hiprand_uniform(&states[ray_index]) < F)
                {
                    new_dir = vector3_reflect(r.d, m);
                    if (wiDotN * vector3_dot(new_dir, min.normal) <= 0.0f)
                    {
                        ray_statuses[index] = -1;
                        return;
                    }
                    vector3_add_to(&new_origin, vector3_mul(norm_o, ray_bias));
                }
                else
                {
                    float eta = wiDotM < 0.0f ? min.m->ior : 1.0f / min.m->ior;
                    ray_mediums[ray_index] = wiDotM > 0.0f ? min.m->medium : medium_air();
                    new_dir = vector3_sub(
                        vector3_mul(m, wiDotM * eta - (wiDotM > 0.0f ? 1.0f : -1.0f) * wiDotT),
                        vector3_mul(wi, eta));
                    if (wiDotN * vector3_dot(new_dir, min.normal) >= 0.0f)
                    {
                        ray_statuses[index] = -1;
                        return;
                    }
                    vector3_add_to(&new_origin, vector3_mul(norm_o, -ray_bias));
                }

                float G = Microfacet::G_Beckmann(wi, new_dir, m, a);
                float weight = (G * fabsf(wiDotM)) / (fabsf(wiDotN) * fabsf(vector3_dot(m, min.normal)));
                vector3_mul_vector_to(&ray_colors[ray_index], vector3_mul(albedo, weight));
            }
            else if (min.m->type == MATERIAL_CONDUCTOR)
            {
                // albedo used to store eta
                float cosI = -vector3_dot(r.d, min.normal);
                if (cosI <= 0)
                {
                    ray_statuses[index] = -1;
                    return;
                }
                new_dir = vector3_reflect(r.d, norm_o);
                vector3_add_to(&new_origin, vector3_mul(norm_o, ray_bias));
                vector3_mul_vector_to(&ray_colors[ray_index], Fresnel::conductor(min.m->eta, min.m->k, cosI));
            }
            else if (min.m->type == MATERIAL_ROUGHCONDUCTOR)
            {
                Vector3 wi = vector3_mul(r.d, -1.0f);
                float wiDotN = vector3_dot(wi, min.normal);

                if (wiDotN <= 0)
                {
                    ray_statuses[index] = -1;
                    return;
                }
                float a = min.m->roughness * (1.2f - 0.2f * sqrtf(fabsf(wiDotN)));

                float u1 = hiprand_uniform(&states[ray_index]);
                float u2 = hiprand_uniform(&states[ray_index]);
                Vector3 m = Microfacet::sample_Beckmann(a, u1, u2);
                m = vector3_to_basis(m, min.normal);

                Vector3 F = Fresnel::conductor(min.m->eta, min.m->k, wiDotN);

                new_dir = vector3_reflect(r.d, m);
                if (wiDotN * vector3_dot(new_dir, min.normal) <= 0.0f)
                {
                    ray_statuses[index] = -1;
                    return;
                }
                float wiDotM = vector3_dot(wi, m);
                float G = Microfacet::G_Beckmann(wi, new_dir, m, a);
                float weight = (G * fabsf(wiDotM)) / (fabsf(wiDotN) * fabsf(vector3_dot(m, min.normal)));
                vector3_mul_vector_to(&ray_colors[ray_index], vector3_mul(F, weight));
                vector3_add_to(&new_origin, vector3_mul(m, ray_bias));
            }
            else if (min.m->type == MATERIAL_PLASTIC)
            {
                // Simplified model based on Mitsuba - removed weighting towards diffuse or specular samples
                float cosI = -vector3_dot(r.d, min.normal);
                float cosT = 0.0f;
                float F = Fresnel::dielectric(cosI, min.m->ior, cosT);
                float u1  = hiprand_uniform(&states[ray_index]);
                if (F == 1.0f || u1 < F)
                {
                    new_dir = vector3_reflect(r.d, min.normal);
                    vector3_mul_vector_to(&ray_colors[ray_index], vector3_create(0.99, 0.99, 0.99));
                }
                else
                {
                    float u2 = hiprand_uniform(&states[ray_index]);
                    new_dir = vector3_to_basis(sample_hemisphere_cosine((u1 - F) / (1.0 - F), u2), norm_o);
                    float cosO = vector3_dot(new_dir, min.normal);
                    float Fo = Fresnel::dielectric(cosO, min.m->ior, cosT);
                    // fresnel integral stored in roughness
                    Vector3 diff = vector3_div(albedo, 1.0f - min.m->roughness);
                    float inv_eta = 1.0f / min.m->ior;
                    vector3_mul_vector_to(&ray_colors[ray_index], vector3_mul(diff, inv_eta * inv_eta * (1 - Fo)));
                }

                vector3_add_to(&new_origin, vector3_mul(norm_o, ray_bias));
            }

			ray_set(&rays[ray_index], new_origin, new_dir);
		}
		else
		{
            float u = atan2f(rays[ray_index].d.z, rays[ray_index].d.x) * 0.5f * ILLUME_INV_PI + 0.5f;
            float v = 0.5f - asinf(rays[ray_index].d.y) * ILLUME_INV_PI;

			vector3_mul_vector_to(&ray_colors[ray_index], scene->envmap.eval(Vec2f(u, v)));
			vector3_add_to(&final_colors[ray_index], ray_colors[ray_index]);
			ray_statuses[index] = -1;
		}
	}
}

static void compact_pixels(int* d_ray_statuses, int* h_ray_statuses, int* active_pixels)
{
	int pixels = *active_pixels;
	int size = pixels * sizeof(int); 
	HANDLE_ERROR( hipMemcpy(h_ray_statuses, d_ray_statuses, size, hipMemcpyDeviceToHost) );

	int left = 0;
	int right = pixels - 1;
	while (left < right)
	{
		while (h_ray_statuses[left] != -1 && left < pixels)
		{
			left++;
		}
		while (h_ray_statuses[right] == -1 && right >= 0)
		{
			right--;
		}

		if (left < right)
		{
			h_ray_statuses[left] = h_ray_statuses[right];
			h_ray_statuses[right] = -1;
		}
	}

	left = 0;
	while (left < pixels && h_ray_statuses[left] != -1)
	{
		left++;
	}
	*active_pixels = left;

	HANDLE_ERROR( hipMemcpy(d_ray_statuses, h_ray_statuses, size, hipMemcpyHostToDevice) );
}

__global__
void tonemap(Vector3* final_colors, Pixel* pixels, float samples, Tonemapper op, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
	{
		Vector3 avg = vector3_mul(final_colors[index], 1 / samples);
		//avg = vector3_max(vector3_min(corrected, 1), 0);
		//corrected = vector3_pow(corrected, gamma);
        avg = op.eval(avg);
        avg = vector3_max(vector3_min(avg, 1), 0);
		pixels[index].red = (int) (255 * avg.x);
		pixels[index].green = (int) (255 * avg.y);
		pixels[index].blue = (int) (255 * avg.z);
	}
}

static RenderInfo* allocate_render_info_gpu(int width, int height, Camera& camera)
{
	RenderInfo i;
	i.image_width = width;
	float dim_ratio = (float) height / (float) width;
	float tan_half_fov = tanf(ILLUME_PI * camera.fov / 360);
	i.camera = camera;
	float dofmfov = camera.dof * tan_half_fov;
	i.camera_pixel_size = dofmfov * 2 / (float) width;
	i.camera_left = -1 * dofmfov;
	i.camera_top = dim_ratio * dofmfov;
	RenderInfo *d_info;
	HANDLE_ERROR( hipMalloc(&d_info, sizeof(RenderInfo)) );
	HANDLE_ERROR( hipMemcpy(d_info, &i, sizeof(RenderInfo), hipMemcpyHostToDevice) );
	return d_info;
}

static Vector3* allocate_final_colors_gpu(int pixels_amount)
{
	Vector3* h_final_colors = (Vector3 *) malloc(sizeof(Vector3) * pixels_amount);
	for (int i = 0; i < pixels_amount; i++)
	{
		h_final_colors[i] = vector3_create(0, 0, 0);
	}
	Vector3* d_final_colors;
	HANDLE_ERROR( hipMalloc(&d_final_colors, pixels_amount * sizeof(Vector3)) );
	HANDLE_ERROR( hipMemcpy(d_final_colors, h_final_colors, pixels_amount * sizeof(Vector3), hipMemcpyHostToDevice) );
	free(h_final_colors);
	return d_final_colors;
}

static void start_timer(hipEvent_t* start, hipEvent_t* stop)
{
	HANDLE_ERROR( hipEventCreate(start) );
	HANDLE_ERROR( hipEventCreate(stop) );
	HANDLE_ERROR( hipEventRecord(*start, 0) );
}

static void end_timer(hipEvent_t* start, hipEvent_t* stop, float* time)
{
	HANDLE_ERROR( hipEventRecord(*stop, 0) );
	HANDLE_ERROR( hipEventSynchronize(*stop) );
	HANDLE_ERROR( hipEventElapsedTime(time, *start, *stop) );
}

uint32_t wang_hash(uint32_t a)
{
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

void Renderer::render_to_bitmap(Bitmap* bitmap)
{
    printf("%zu\n", sizeof(Material));
	hipEvent_t render_start;
	hipEvent_t render_stop;
	start_timer(&render_start, &render_stop);

	HANDLE_ERROR( hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 1024) );
	int pixels_amount = m_width * m_height;
	int threads_per_block = 256;
	int blocks_amount = (pixels_amount + threads_per_block - 1) / threads_per_block;

	hiprandState* d_states;
	HANDLE_ERROR( hipMalloc(&d_states, sizeof(hiprandState) * threads_per_block * blocks_amount) );

	RenderInfo* d_info = 
		allocate_render_info_gpu(m_width, m_height, m_scene.get_camera());

	Vector3* d_final_colors = allocate_final_colors_gpu(pixels_amount);

	Vector3* d_ray_colors;
	HANDLE_ERROR( hipMalloc(&d_ray_colors, pixels_amount * sizeof(Vector3)) );

	int* d_ray_statuses;
	HANDLE_ERROR( hipMalloc(&d_ray_statuses, pixels_amount * sizeof(int)) );

	Medium* d_ray_mediums;
	HANDLE_ERROR( hipMalloc(&d_ray_mediums, pixels_amount * sizeof(Medium)) );

	Ray* d_rays;
	HANDLE_ERROR( hipMalloc(&d_rays, sizeof(Ray) * pixels_amount) );
    SceneRef device_scene(m_scene);

	int* h_ray_statuses = (int *) calloc(pixels_amount, sizeof(int));

	printf("Rendering...    "); fflush(stdout);
	int last_progress = -1;
	float progress_step = 100.0f / (float) m_spp;
	hipEvent_t start, stop;
	for (int i = 0; i < m_spp; i++)
	{
		start_timer(&start, &stop);
		init_curand_states KERNEL_ARGS2(blocks_amount, threads_per_block) (d_states, wang_hash(i), pixels_amount);

		init_rays KERNEL_ARGS2(blocks_amount, threads_per_block)
			(d_rays, d_ray_statuses, d_ray_colors, d_ray_mediums, d_info, d_states, pixels_amount);

		int active_pixels = pixels_amount;
		int blocks = blocks_amount;

		for (int j = 0; j < m_max_depth; j++)
		{
			pathtrace_kernel KERNEL_ARGS2(blocks, threads_per_block)
				(d_final_colors, d_rays, d_ray_statuses, d_ray_colors, d_ray_mediums,
				 j, device_scene.getScene(), d_states, m_ray_bias, active_pixels);
			compact_pixels(d_ray_statuses, h_ray_statuses, &active_pixels);
			blocks = (active_pixels + threads_per_block - 1) / threads_per_block;
		}
		int progress = (int) ((float) i * progress_step);
		if (progress != last_progress)
		{
			printf("\b\b\b%02d%%", progress); fflush(stdout);
			last_progress = progress;
		}
		
	}
	printf("\b\b\b100%%\n");

	HANDLE_ERROR( hipFree(d_states) );
	HANDLE_ERROR( hipFree(d_rays) );
	HANDLE_ERROR( hipFree(d_info) );
	HANDLE_ERROR( hipFree(d_ray_statuses) );
	HANDLE_ERROR( hipFree(d_ray_colors) );
	HANDLE_ERROR( hipFree(d_ray_mediums) );
	free(h_ray_statuses);

	Pixel* d_pixels;
	HANDLE_ERROR( hipMalloc(&d_pixels, sizeof(Pixel) * pixels_amount) );
	HANDLE_ERROR( hipMemcpy(d_pixels, bitmap->pixels, sizeof(Pixel) * pixels_amount, hipMemcpyHostToDevice) );

	tonemap KERNEL_ARGS2(blocks_amount, threads_per_block) 
		(d_final_colors, d_pixels, (float) m_spp, m_tonemapper, pixels_amount);
	HANDLE_ERROR( hipMemcpy(bitmap->pixels, d_pixels, sizeof(Pixel) * pixels_amount, hipMemcpyDeviceToHost) );

	HANDLE_ERROR( hipFree(d_final_colors) );
	HANDLE_ERROR( hipFree(d_pixels) );

	float render_time;
	end_timer(&render_start, &render_stop, &render_time);

	printf("Render time: %f seconds\n", 1e-3 * (double) render_time);
}

int Renderer::get_width()
{
	return m_width;
}

int Renderer::get_height()
{
	return m_height;
}