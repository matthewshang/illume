#include "hip/hip_runtime.h"
// Header files that include cuda code here to avoid C compiler issues

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "kernel.h"

#include "vector3.h"
#include "ray.h"
#include "sphere.h"
#include "sample.h"
#include "scene.h"

__global__ 
void init_curand_states(hiprandState* states, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		hiprand_init(666420691337, index, 0, &states[index]);
	}
}

__global__
void init_rays(Ray* rays, int* ray_statuses, Vector3* ray_colors, RenderInfo* info, hiprandState* states, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		int x = index % (int) info->image_width;
		int y = index / (int) info->image_width;
		float left_edge = info->camera_left + info->camera_pixel_size * (float) x;
		float right_edge = left_edge + info->camera_pixel_size;
		float top_edge = info->camera_top - info->camera_pixel_size * (float) y;
		float bottom_edge = top_edge + info->camera_pixel_size;

		float r_x = left_edge + (right_edge - left_edge) * hiprand_uniform(&states[index]);
		float r_y = bottom_edge + (top_edge - bottom_edge) * hiprand_uniform(&states[index]);

		rays[index] = ray_create(vector3_create(0, 0, 0), vector3_create(r_x, r_y, info->camera_focus_plane));
		ray_statuses[index] = index;
		ray_colors[index] = vector3_create(1, 1, 1);
	}
}

__global__
void pathtrace_kernel(Vector3* final_colors, Ray* rays, int* ray_statuses, 
					  Vector3* ray_colors, Scene* scene, hiprandState* states, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int ray_index = ray_statuses[index];
	if (index < N && ray_index != -1)
	{
		Intersection min = intersection_create_no_intersect();
		min.d = FLT_MAX;
		for (int i = 0; i < scene->sphere_amount; i++)
		{
			Intersection inter = sphere_ray_intersection(&scene->spheres[i], &rays[ray_index]);

			if (inter.is_intersect == 1 && inter.d < min.d)
			{
				min = inter;
			}
		}
		if (min.is_intersect == 1)
		{
			Vector3 red = vector3_create(255, 0, 0);
			vector3_mul_vector_to(&ray_colors[ray_index], &red);
			Vector3 new_origin = ray_position_along(&rays[ray_index], min.d);
			Vector3 bias = vector3_mul(&min.normal, 0.00001);
			vector3_add_to(&new_origin, &bias);
			float u1 = hiprand_uniform(&states[ray_index]);
			float u2 = hiprand_uniform(&states[ray_index]);
			Vector3 sample = sample_hemisphere_cosine(u1, u2);
			Vector3 new_direction = vector3_to_basis(&sample, &min.normal);
			ray_set(&rays[ray_index], new_origin, new_direction);
		}
		else
		{
			Vector3 blue = vector3_create(135, 206, 235);
			vector3_mul_vector_to(&ray_colors[ray_index], &blue);
			vector3_add_to(&final_colors[ray_index], &ray_colors[ray_index]);
			ray_statuses[ray_index] = -1;
		}
	}
}

__global__
void set_bitmap(Vector3* final_colors, Pixel* pixels, float samples, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
	{
		pixels[index].red = (int) (final_colors[index].x / samples);
		pixels[index].green = (int) (final_colors[index].y / samples);
		pixels[index].blue = (int) (final_colors[index].z / samples);
	}
}

static void init_render_info(RenderInfo* i, int width, int height, float fov, float plane)
{
	i->image_width = width;
	float dim_ratio = (float) height / (float) width;
	float tan_half_fov = tanf(PI * fov / 360);
	i->camera_focus_plane = plane;	
	i->camera_pixel_size = tan_half_fov * 2 / (float) width;
	i->camera_left = -1 * plane * tan_half_fov;
	i->camera_top = dim_ratio * plane * tan_half_fov;
}

void render_scene(Bitmap* bitmap, int samples)
{
	struct timespec tstart = {0, 0};
	struct timespec tend = {0, 0};
	clock_gettime(CLOCK_MONOTONIC, &tstart);

	Scene* scene = scene_new(2);
	scene->spheres[0] = sphere_create(1, vector3_create(0, 0, 4));
	scene->spheres[1] = sphere_create(1, vector3_create(2, 0, 4));

	hipDeviceSetLimit(hipLimitMallocHeapSize, 256 * 1024 * 1024);
	int N = bitmap->width * bitmap->height;
	int threads_per_block = 256;
	int blocks_amount = (N + threads_per_block - 1) / threads_per_block;

	RenderInfo info;
	init_render_info(&info, bitmap->width, bitmap->height, 90, 1);
	RenderInfo* d_info;
	hipMalloc(&d_info, sizeof(RenderInfo));
	hipMemcpy(d_info, &info, sizeof(RenderInfo), hipMemcpyHostToDevice);

	hiprandState* d_states;
	hipMalloc(&d_states, sizeof(hiprandState) * threads_per_block * blocks_amount);
	init_curand_states<<<blocks_amount, threads_per_block>>>(d_states, N);

	Vector3* h_final_colors = (Vector3 *) malloc(sizeof(Vector3) * N);
	for (int i = 0; i < N; i++)
	{
		h_final_colors[i] = vector3_create(0, 0, 0);
	}
	Vector3* d_final_colors;
	hipMalloc(&d_final_colors, N * sizeof(Vector3));
	hipMemcpy(d_final_colors, h_final_colors, N * sizeof(Vector3), hipMemcpyHostToDevice);

	Vector3* d_ray_colors;
	hipMalloc(&d_ray_colors, N * sizeof(Vector3));

	int* d_ray_statuses;
	hipMalloc(&d_ray_statuses, N * sizeof(int));

	Ray* d_rays;
	hipMalloc(&d_rays, sizeof(Ray) * N);

	int spheres_size = sizeof(Sphere) * scene->sphere_amount;
	Scene* d_scene;
	hipMalloc(&d_scene, sizeof(Scene));
	Sphere* d_spheres;
	hipMalloc(&d_spheres, spheres_size);
	Sphere* h_spheres = scene->spheres;
	scene->spheres = d_spheres;
	hipMemcpy(d_scene, scene, sizeof(Scene), hipMemcpyHostToDevice);
	scene->spheres = h_spheres;

	hipMemcpy(d_spheres, scene->spheres, spheres_size, hipMemcpyHostToDevice);

	struct timespec tstart_render = {0, 0};
	struct timespec tend_render = {0, 0};
	clock_gettime(CLOCK_MONOTONIC, &tstart_render);

	for (int i = 0; i < samples; i++)
	{
		init_rays<<<blocks_amount, threads_per_block>>>(d_rays, d_ray_statuses, d_ray_colors, d_info, d_states, N);

		for (int j = 0; j < 5; j++)
		{
			pathtrace_kernel<<<blocks_amount, threads_per_block>>>(d_final_colors, d_rays, d_ray_statuses, d_ray_colors, d_scene, d_states, N);		
		}
	}

	clock_gettime(CLOCK_MONOTONIC, &tend_render);
	printf("Render Time: %f seconds\n", 
		    ((double) tend_render.tv_sec + 1.0e-9 * tend_render.tv_nsec) -
		    ((double) tstart_render.tv_sec + 1.0e-9 * tstart_render.tv_nsec));


	hipFree(d_states);
	hipFree(d_rays);
	hipFree(d_info);
	hipFree(d_ray_statuses);
	hipFree(d_ray_colors);
	hipFree(d_spheres);
	hipFree(d_scene);
	scene_free(scene);


	Pixel* h_pixels = bitmap->pixels;
	Pixel* d_pixels;
	hipMalloc(&d_pixels, sizeof(Pixel) * N);
	hipMemcpy(d_pixels, h_pixels, sizeof(Pixel) * N, hipMemcpyHostToDevice);

	set_bitmap<<<blocks_amount, threads_per_block>>>(d_final_colors, d_pixels, (float) samples, N);
	hipMemcpy(h_pixels, d_pixels, sizeof(Pixel) * N, hipMemcpyDeviceToHost);

	hipFree(d_final_colors);
	free(h_final_colors);
	hipFree(d_pixels);


	clock_gettime(CLOCK_MONOTONIC, &tend);
	printf("Total Time: %f seconds\n", 
		    ((double) tend.tv_sec + 1.0e-9 * tend.tv_nsec) -
		    ((double) tstart.tv_sec + 1.0e-9 * tstart.tv_nsec));

}