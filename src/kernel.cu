#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ 
void init_curand_states(hiprandState* states, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		hiprand_init((666420691337 << 20) + index, 0, 0, &states[index]);
	}
}

typedef struct
{
	float image_width;
	float camera_dof;
	float camera_aperture;
	Vector3 camera_pos;
	float camera_pixel_size;
	float camera_left;
	float camera_top;
} 
RenderInfo;

__global__
void init_rays(Ray* rays, int* ray_statuses, Vector3* ray_colors, RenderInfo* info, hiprandState* states, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		RenderInfo i = *info;
		int x = index % (int) i.image_width;
		int y = index / (int) i.image_width;
		float left_edge = i.camera_left + i.camera_pixel_size * (float) x;
		float right_edge = left_edge + i.camera_pixel_size;
		float top_edge = i.camera_top - i.camera_pixel_size * (float) y;
		float bottom_edge = top_edge + i.camera_pixel_size;

		float r_x = left_edge + (right_edge - left_edge) * hiprand_uniform(&states[index]);
		float r_y = bottom_edge + (top_edge - bottom_edge) * hiprand_uniform(&states[index]);

		Vector3 pos;
		if (i.camera_aperture == 0)
		{
			pos = i.camera_pos;
		}
		else
		{
			float u1 = hiprand_uniform(&states[index]);
			float u2 = hiprand_uniform(&states[index]);
			pos = 
				vector3_add(vector3_mul(sample_circle(u1, u2), i.camera_aperture), i.camera_pos);
		}
		rays[index] = ray_create(pos, vector3_sub(vector3_create(r_x, r_y, i.camera_dof), pos));
		ray_statuses[index] = index;
		ray_colors[index] = vector3_create(1, 1, 1);
	}
}

__device__
static Intersection get_min_intersection(Scene* scene, Ray ray)
{
	Intersection min = intersection_create_no_intersect();
	min.d = FLT_MAX;
	for (int i = 0; i < scene->sphere_amount; i++)
	{
		Intersection inter = sphere_ray_intersect(scene->spheres[i], ray);

		if (inter.is_intersect == 1 && inter.d < min.d)
		{
			min = inter;
		}
	}

	for (int i = 0; i < scene->plane_amount; i++)
	{
		Intersection inter = plane_ray_intersect(scene->planes[i], ray);

		if (inter.is_intersect == 1 && inter.d < min.d)
		{
			min = inter;
		}
	}

	for (int i = 0; i < scene->mesh_amount; i++)
	{
		Intersection inter = mesh_ray_intersect(scene->meshes[i], ray);

		if (inter.is_intersect == 1 && inter.d < min.d)
		{
			min = inter;
		}
	}

	return min;
}

__device__
static Vector3 get_background_color(Vector3 direction)
{
	Vector3 sun = vector3_create(1, 1, -1);
	vector3_normalize(&sun);
	float grad = (vector3_dot(direction, sun) + 1) / 2;
	return vector3_add(vector3_mul(vector3_create(0.2, 0.2, 0.2), 1 - grad), 
					   vector3_mul(vector3_create(0.8, 0.8, 0.8), grad));
}

__global__
void pathtrace_kernel(Vector3* final_colors, Ray* rays, int* ray_statuses, 
					  Vector3* ray_colors, Scene* scene, hiprandState* states, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int ray_index = ray_statuses[index];
	if (index < N && ray_index != -1)
	{
		Intersection min = get_min_intersection(scene, rays[ray_index]);
		if (min.is_intersect == 1)
		{
			if (min.m.type == MATERIAL_EMISSIVE)
			{
				vector3_mul_vector_to(&ray_colors[ray_index], min.m.c);
				vector3_add_to(&final_colors[ray_index], ray_colors[ray_index]);
				ray_statuses[index] = -1;
			}
			else if (min.m.type == MATERIAL_DIFFUSE)
			{
				vector3_mul_vector_to(&ray_colors[ray_index], min.m.c);
				Vector3 new_origin = ray_position_along(rays[ray_index], min.d);
				vector3_add_to(&new_origin, vector3_mul(min.normal, 10e-6));
				float u1 = hiprand_uniform(&states[ray_index]);
				float u2 = hiprand_uniform(&states[ray_index]);
				Vector3 sample = sample_hemisphere_cosine(u1, u2);
				Vector3 new_direction = vector3_to_basis(sample, min.normal);
				ray_set(&rays[ray_index], new_origin, new_direction);
			}
			else
			{
				Ray r = rays[ray_index];
				vector3_mul_vector_to(&ray_colors[ray_index], min.m.c);
				Vector3 new_origin = ray_position_along(r, min.d);
				vector3_add_to(&new_origin, vector3_mul(min.normal, 10e-6));
				Vector3 new_direction = vector3_reflect(r.d, min.normal);
				ray_set(&rays[ray_index], new_origin, new_direction);
			}
		}
		else
		{
			Vector3 sky = get_background_color(rays[ray_index].d);
			vector3_mul_vector_to(&ray_colors[ray_index], sky);
			vector3_add_to(&final_colors[ray_index], ray_colors[ray_index]);
			ray_statuses[index] = -1;
		}
	}
}

static void compact_pixels(int* d_ray_statuses, int* h_ray_statuses, int* active_pixels)
{
	int pixels = *active_pixels;
	int size = pixels * sizeof(int); 
	HANDLE_ERROR( hipMemcpy(h_ray_statuses, d_ray_statuses, size, hipMemcpyDeviceToHost) );
	
	int left = 0;
	int right = pixels - 1;
	while (left < right)
	{
		while (h_ray_statuses[left] != -1 && left < pixels)
		{
			left++;
		}
		while (h_ray_statuses[right] == -1 && right >= 0)
		{
			right--;
		}
		if (left < right)
		{
			h_ray_statuses[left] = h_ray_statuses[right];
			h_ray_statuses[right] = -1;
			*active_pixels = left;
		}
	}

	HANDLE_ERROR( hipMemcpy(d_ray_statuses, h_ray_statuses, size, hipMemcpyHostToDevice) );
}

__global__
void set_bitmap(Vector3* final_colors, Pixel* pixels, float samples, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
	{
		float gamma = 1 / 2.2;
		Vector3 corrected = vector3_mul(final_colors[index], 1 / samples);
		corrected = vector3_max(vector3_min(corrected, 1), 0);
		corrected = vector3_pow(corrected, gamma);
		pixels[index].red = (int) (255 * corrected.x);
		pixels[index].green = (int) (255 * corrected.y);
		pixels[index].blue = (int) (255 * corrected.z);
	}
}

static RenderInfo* allocate_render_info_gpu(int width, int height, Camera camera)
{
	RenderInfo i;
	i.image_width = width;
	float dim_ratio = (float) height / (float) width;
	float tan_half_fov = tanf(PI * camera.fov / 360);
	i.camera_dof = camera.dof;	
	i.camera_aperture = camera.aperture;
	i.camera_pos = camera.pos;
	float dofmfov = i.camera_dof * tan_half_fov;
	i.camera_pixel_size = dofmfov * 2 / (float) width;
	i.camera_left = -1 * dofmfov;
	i.camera_top = dim_ratio * dofmfov;
	RenderInfo *d_info;
	HANDLE_ERROR( hipMalloc(&d_info, sizeof(RenderInfo)) );
	HANDLE_ERROR( hipMemcpy(d_info, &i, sizeof(RenderInfo), hipMemcpyHostToDevice) );
	return d_info;
}

static Vector3* allocate_final_colors_gpu(int pixels_amount)
{
	Vector3* h_final_colors = (Vector3 *) malloc(sizeof(Vector3) * pixels_amount);
	for (int i = 0; i < pixels_amount; i++)
	{
		h_final_colors[i] = vector3_create(0, 0, 0);
	}
	Vector3* d_final_colors;
	HANDLE_ERROR( hipMalloc(&d_final_colors, pixels_amount * sizeof(Vector3)) );
	HANDLE_ERROR( hipMemcpy(d_final_colors, h_final_colors, pixels_amount * sizeof(Vector3), hipMemcpyHostToDevice) );
	free(h_final_colors);
	return d_final_colors;
}

typedef struct
{
	Scene* d_scene;
	Sphere* d_spheres;
	Plane* d_planes;
	Mesh* d_meshes;
	int mesh_amount;
	Triangle** d_triangle_pointers;
} 
SceneReference;

static SceneReference allocate_scene_gpu(Scene* scene)
{
	SceneReference ref;
	int spheres_size = sizeof(Sphere) * scene->sphere_amount;
	int planes_size = sizeof(Plane) * scene->plane_amount;
	int meshes_size = sizeof(Mesh) * scene->mesh_amount;
	ref.mesh_amount = scene->mesh_amount;

	HANDLE_ERROR( hipMalloc(&ref.d_scene, sizeof(Scene)) );
	HANDLE_ERROR( hipMalloc(&ref.d_spheres, spheres_size) );
	HANDLE_ERROR( hipMalloc(&ref.d_planes, planes_size) );
	HANDLE_ERROR( hipMalloc(&ref.d_meshes, meshes_size) );
	ref.d_triangle_pointers = (Triangle **) calloc(scene->mesh_amount, sizeof(Triangle *));
	for (int i = 0; i < scene->mesh_amount; i++)
	{
		int triangles_size = scene->meshes[i].triangle_amount * sizeof(Triangle);
		HANDLE_ERROR( hipMalloc(&ref.d_triangle_pointers[i], triangles_size) );
		HANDLE_ERROR( hipMemcpy(
			ref.d_triangle_pointers[i], scene->meshes[i].triangles, triangles_size, hipMemcpyHostToDevice) );
	}

	Triangle** h_triangle_pointers = (Triangle **) calloc(scene->mesh_amount, sizeof(Triangle *));
	for (int i = 0; i < scene->mesh_amount; i++)
	{
		h_triangle_pointers[i] = scene->meshes[i].triangles;
		scene->meshes[i].triangles = ref.d_triangle_pointers[i];	
	}

	Sphere* h_spheres = scene->spheres;
	Plane* h_planes = scene->planes;
	Mesh* h_meshes = scene->meshes;
	scene->spheres = ref.d_spheres;
	scene->planes = ref.d_planes;
	scene->meshes = ref.d_meshes;
	HANDLE_ERROR( hipMemcpy(ref.d_scene, scene, sizeof(Scene), hipMemcpyHostToDevice) );
	scene->spheres = h_spheres;
	scene->planes = h_planes;
	scene->meshes = h_meshes;
	HANDLE_ERROR( hipMemcpy(ref.d_spheres, scene->spheres, spheres_size, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(ref.d_planes, scene->planes, planes_size, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(ref.d_meshes, scene->meshes, meshes_size, hipMemcpyHostToDevice) );
	for (int i = 0; i < scene->mesh_amount; i++)
	{
		scene->meshes[i].triangles = h_triangle_pointers[i];
	}
	free(h_triangle_pointers);
	return ref;
}

static void free_scene_gpu(SceneReference ref)
{
	HANDLE_ERROR( hipFree(ref.d_spheres) );
	HANDLE_ERROR( hipFree(ref.d_planes) );
	HANDLE_ERROR( hipFree(ref.d_meshes) );
	HANDLE_ERROR( hipFree(ref.d_scene) );
	for (int i = 0; i < ref.mesh_amount; i++)
	{
		HANDLE_ERROR( hipFree(ref.d_triangle_pointers[i]) );
	}
	free(ref.d_triangle_pointers);
}

static void start_timer(hipEvent_t* start, hipEvent_t* stop)
{
	HANDLE_ERROR( hipEventCreate(start) );
	HANDLE_ERROR( hipEventCreate(stop) );
	HANDLE_ERROR( hipEventRecord(*start, 0) );
}

static void end_timer(hipEvent_t* start, hipEvent_t* stop, float* time)
{
	HANDLE_ERROR( hipEventRecord(*stop, 0) );
	HANDLE_ERROR( hipEventSynchronize(*stop) );
	HANDLE_ERROR( hipEventElapsedTime(time, *start, *stop) );
}

void render_scene(Scene* scene, Bitmap* bitmap, Camera camera, int samples, int max_depth)
{
	if (!scene)
	{
		return;
	}

	hipEvent_t render_start;
	hipEvent_t render_stop;
	start_timer(&render_start, &render_stop);

	HANDLE_ERROR( hipDeviceSetLimit(hipLimitMallocHeapSize, 256 * 1024 * 1024) );
	int pixels_amount = bitmap->width * bitmap->height;
	int threads_per_block = 256;
	int blocks_amount = (pixels_amount + threads_per_block - 1) / threads_per_block;

	hiprandState* d_states;
	HANDLE_ERROR( hipMalloc(&d_states, sizeof(hiprandState) * threads_per_block * blocks_amount) );
	init_curand_states<<<blocks_amount, threads_per_block>>>(d_states, pixels_amount);

	RenderInfo* d_info = 
		allocate_render_info_gpu(bitmap->width, bitmap->height, camera);

	Vector3* d_final_colors = allocate_final_colors_gpu(pixels_amount);

	Vector3* d_ray_colors;
	HANDLE_ERROR( hipMalloc(&d_ray_colors, pixels_amount * sizeof(Vector3)) );

	int* d_ray_statuses;
	HANDLE_ERROR( hipMalloc(&d_ray_statuses, pixels_amount * sizeof(int)) );

	Ray* d_rays;
	HANDLE_ERROR( hipMalloc(&d_rays, sizeof(Ray) * pixels_amount) );

	SceneReference ref = allocate_scene_gpu(scene);

	int* h_ray_statuses = (int *) calloc(pixels_amount, sizeof(int));

	hipEvent_t calc_start;
	hipEvent_t calc_stop;
	start_timer(&calc_start, &calc_stop);

	for (int i = 0; i < samples; i++)
	{
		init_rays<<<blocks_amount, threads_per_block>>>
			(d_rays, d_ray_statuses, d_ray_colors, d_info, d_states, pixels_amount);

		int active_pixels = pixels_amount;
		int blocks = blocks_amount;

		for (int j = 0; j < max_depth; j++)
		{
			pathtrace_kernel<<<blocks, threads_per_block>>>
				(d_final_colors, d_rays, d_ray_statuses, d_ray_colors, 
				 ref.d_scene, d_states, active_pixels);		

			compact_pixels(d_ray_statuses, h_ray_statuses, &active_pixels);
			blocks = (active_pixels + threads_per_block - 1) / threads_per_block;
		}
	}

	float calc_time;
	end_timer(&calc_start, &calc_stop, &calc_time);

	HANDLE_ERROR( hipFree(d_states) );
	HANDLE_ERROR( hipFree(d_rays) );
	HANDLE_ERROR( hipFree(d_info) );
	HANDLE_ERROR( hipFree(d_ray_statuses) );
	HANDLE_ERROR( hipFree(d_ray_colors) );
	free_scene_gpu(ref);
	free(h_ray_statuses);

	Pixel* h_pixels = bitmap->pixels;
	Pixel* d_pixels;
	HANDLE_ERROR( hipMalloc(&d_pixels, sizeof(Pixel) * pixels_amount) );
	HANDLE_ERROR( hipMemcpy(d_pixels, h_pixels, sizeof(Pixel) * pixels_amount, hipMemcpyHostToDevice) );

	set_bitmap<<<blocks_amount, threads_per_block>>>(d_final_colors, d_pixels, (float) samples, pixels_amount);
	HANDLE_ERROR( hipMemcpy(h_pixels, d_pixels, sizeof(Pixel) * pixels_amount, hipMemcpyDeviceToHost) );

	HANDLE_ERROR( hipFree(d_final_colors) );
	HANDLE_ERROR( hipFree(d_pixels) );

	float render_time;
	end_timer(&render_start, &render_stop, &render_time);

	printf("Calculation time: %f seconds\n", 1e-3 * (double) calc_time);
	printf("Render time: %f seconds\n", 1e-3 * (double) render_time);
}