#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "kernel.h"
#include "bitmap.h"
#include "vector3.h"
#include "ray.h"
#include "sphere.h"

__global__ 
void init_curand_states(hiprandState* states, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		hiprand_init(666420691337, index, 0, &states[index]);
	}
}

__global__
void init_rays(Ray* rays, RenderInfo* info, hiprandState* states, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		int x = index % (int) info->image_width;
		int y = index / (int) info->image_height;
		float left_edge = 
			info->camera_left + ((float) x / info->image_width) * info->camera_width;
		float right_edge = 
			info->camera_left + ((float) (x + 1) / info->image_width) * info->camera_width;
		float top_edge = 
			info->camera_top - ((float) y / info->image_height) * info->camera_height;
		float bottom_edge = 
			info->camera_top - ((float) (y + 1) / info->image_height) * info->camera_height;

		float r_x = left_edge + (right_edge - left_edge) * hiprand_uniform(&states[index]);
		float r_y = bottom_edge + (top_edge - bottom_edge) * hiprand_uniform(&states[index]);

		rays[index] = ray_create(vector3_create(0, 0, 0), vector3_create(r_x, r_y, 1));
	}
}

__global__
void pathtrace_kernel(Vector3* colors, Ray* rays, Sphere* sphere, hiprandState* states, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
	{
		if (sphere_ray_intersection(sphere, &rays[index]).is_intersect == 1)
		{
			Vector3 red = vector3_create(255, 0, 0);
			vector3_add_to(&colors[index], &red);
		}
		else
		{
			Vector3 blue = vector3_create(135, 206, 235);
			vector3_add_to(&colors[index], &blue);
		}
	}
}

__global__
void set_bitmap(Vector3* colors, Pixel* pixels, float samples, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
	{
		pixels[index].red = (int) (colors[index].x / samples);
		pixels[index].green = (int) (colors[index].y / samples);
		pixels[index].blue = (int) (colors[index].z / samples);
	}
}

static void init_render_info(RenderInfo* i, int width, int height, float fov, float plane)
{
	i->image_width = width;
	i->image_height = height;
	i->image_dim_ratio = (float) width / (float) height;
	i->camera_tan_half_fov = tanf(PI * fov / 360);
	i->camera_focus_plane = plane;
	i->camera_width = 2 * plane * i->camera_tan_half_fov;
	i->camera_height = i->camera_width / i->image_dim_ratio;
	i->camera_left = i->camera_width / -2;
	i->camera_top = i->camera_height / 2;
}

void call_kernel(Bitmap* bitmap)
{
	int N = bitmap->width * bitmap->height;
	int threads_per_block = 256;
	int blocks_amount = (N + threads_per_block - 1) / threads_per_block;

	RenderInfo info;
	init_render_info(&info, bitmap->width, bitmap->height, 90, 1);
	RenderInfo* d_info;
	hipMalloc(&d_info, sizeof(RenderInfo));
	hipMemcpy(d_info, &info, sizeof(RenderInfo), hipMemcpyHostToDevice);

	hiprandState* d_states;
	hipMalloc(&d_states, sizeof(hiprandState) * threads_per_block * blocks_amount);
	init_curand_states<<<blocks_amount, threads_per_block>>>(d_states, N);

	Ray* d_rays;
	hipMalloc(&d_rays, sizeof(Ray) * N);

	Sphere* sphere = sphere_new(1, vector3_create(0, 0, 5));
	Sphere* d_sphere;
	hipMalloc(&d_sphere, sizeof(Sphere));
	hipMemcpy(d_sphere, sphere, sizeof(Sphere), hipMemcpyHostToDevice);

	Vector3 h_colors[N];
	for (int i = 0; i < N; i++)
	{
		h_colors[i] = vector3_create(0, 0, 0);
	}
	Vector3* d_colors;
	hipMalloc(&d_colors, N * sizeof(Vector3));
	hipMemcpy(d_colors, &h_colors, N * sizeof(Vector3), hipMemcpyHostToDevice);

	for (int i = 0; i < 50; i++)
	{
		init_rays<<<blocks_amount, threads_per_block>>>(d_rays, d_info, d_states, N);
		pathtrace_kernel<<<blocks_amount, threads_per_block>>>(d_colors, d_rays, d_sphere, d_states, N);		
	}

	Pixel* h_pixels = bitmap->pixels;
	Pixel* d_pixels;
	hipMalloc(&d_pixels, sizeof(Pixel) * N);
	hipMemcpy(d_pixels, h_pixels, sizeof(Pixel) * N, hipMemcpyHostToDevice);

	set_bitmap<<<blocks_amount, threads_per_block>>>(d_colors, d_pixels, 50, N);

	hipMemcpy(h_pixels, d_pixels, sizeof(Pixel) * N, hipMemcpyDeviceToHost);

	hipFree(d_colors);
	hipFree(d_states);
	hipFree(d_sphere);
	hipFree(d_rays);
	hipFree(d_info);
	hipFree(d_pixels);

	sphere_free(sphere);
}