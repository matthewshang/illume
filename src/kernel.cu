#include "hip/hip_runtime.h"
// Header files that include cuda code here to avoid C compiler issues

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "kernel.h"

#include "vector3.h"
#include "ray.h"
#include "sphere.h"
#include "sample.h"
#include "scene.h"
#include "material.h"

__global__ 
void init_curand_states(hiprandState* states, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		hiprand_init(666420691337, index, 0, &states[index]);
	}
}

typedef struct
{
	float image_width;
	float camera_focus_plane;
	float camera_pixel_size;
	float camera_left;
	float camera_top;
} 
RenderInfo;

__global__
void init_rays(Ray* rays, int* ray_statuses, Vector3* ray_colors, RenderInfo* info, hiprandState* states, int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		int x = index % (int) info->image_width;
		int y = index / (int) info->image_width;
		float left_edge = info->camera_left + info->camera_pixel_size * (float) x;
		float right_edge = left_edge + info->camera_pixel_size;
		float top_edge = info->camera_top - info->camera_pixel_size * (float) y;
		float bottom_edge = top_edge + info->camera_pixel_size;

		float r_x = left_edge + (right_edge - left_edge) * hiprand_uniform(&states[index]);
		float r_y = bottom_edge + (top_edge - bottom_edge) * hiprand_uniform(&states[index]);

		rays[index] = ray_create(vector3_create(0, 0, 0), vector3_create(r_x, r_y, info->camera_focus_plane));
		ray_statuses[index] = index;
		ray_colors[index] = vector3_create(1, 1, 1);
	}
}

__device__
static Intersection get_min_intersection(Scene* scene, Ray* ray)
{
	Intersection min = intersection_create_no_intersect();
	min.d = FLT_MAX;
	for (int i = 0; i < scene->sphere_amount; i++)
	{
		Intersection inter = sphere_ray_intersection(&scene->spheres[i], ray);

		if (inter.is_intersect == 1 && inter.d < min.d)
		{
			min = inter;
		}
	}
	return min;
}

__device__
static Vector3 get_background_color(Vector3 direction)
{
	float grad = (direction.x + 2) / 3;
	return vector3_create(grad, grad, grad);
	// return vector3_create(0.8, 0.8, 0.8);
}

__global__
void pathtrace_kernel(Vector3* final_colors, Ray* rays, int* ray_statuses, 
					  Vector3* ray_colors, Scene* scene, hiprandState* states, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int ray_index = ray_statuses[index];
	if (index < N && ray_index != -1)
	{
		Intersection min = get_min_intersection(scene, &rays[ray_index]);
		if (min.is_intersect == 1)
		{
			if (vector3_length2(min.m.e) > 0)
			{
				vector3_mul_vector_to(&ray_colors[ray_index], min.m.e);
				vector3_add_to(&final_colors[ray_index], ray_colors[ray_index]);
				ray_statuses[ray_index] = -1;
			}
			else
			{
				vector3_mul_vector_to(&ray_colors[ray_index], min.m.d);
				Vector3 new_origin = ray_position_along(rays[ray_index], min.d);
				Vector3 bias = vector3_mul(min.normal, 10e-4);
				vector3_add_to(&new_origin, bias);
				float u1 = hiprand_uniform(&states[ray_index]);
				float u2 = hiprand_uniform(&states[ray_index]);
				Vector3 sample = sample_hemisphere_cosine(u1, u2);
				Vector3 new_direction = vector3_to_basis(sample, min.normal);
				ray_set(&rays[ray_index], new_origin, new_direction);
			}
		}
		else
		{
			Vector3 sky = get_background_color(rays[ray_index].d);
			vector3_mul_vector_to(&ray_colors[ray_index], sky);
			vector3_add_to(&final_colors[ray_index], ray_colors[ray_index]);
			ray_statuses[ray_index] = -1;
		}
	}
}

__global__
void set_bitmap(Vector3* final_colors, Pixel* pixels, float samples, int N)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < N)
	{
		pixels[index].red = (int) (255 * final_colors[index].x / samples);
		pixels[index].green = (int) (255 * final_colors[index].y / samples);
		pixels[index].blue = (int) (255 * final_colors[index].z / samples);
	}
}

static RenderInfo* allocate_render_info_gpu(int width, int height, float fov, float plane)
{
	RenderInfo i;
	i.image_width = width;
	float dim_ratio = (float) height / (float) width;
	float tan_half_fov = tanf(PI * fov / 360);
	i.camera_focus_plane = plane;	
	i.camera_pixel_size = tan_half_fov * 2 / (float) width;
	i.camera_left = -1 * plane * tan_half_fov;
	i.camera_top = dim_ratio * plane * tan_half_fov;
	RenderInfo *d_info;
	hipMalloc(&d_info, sizeof(RenderInfo));
	hipMemcpy(d_info, &i, sizeof(RenderInfo), hipMemcpyHostToDevice);
	return d_info;
}

static Vector3* allocate_final_colors_gpu(int pixels_amount)
{
	Vector3* h_final_colors = (Vector3 *) malloc(sizeof(Vector3) * pixels_amount);
	for (int i = 0; i < pixels_amount; i++)
	{
		h_final_colors[i] = vector3_create(0, 0, 0);
	}
	Vector3* d_final_colors;
	hipMalloc(&d_final_colors, pixels_amount * sizeof(Vector3));
	hipMemcpy(d_final_colors, h_final_colors, pixels_amount * sizeof(Vector3), hipMemcpyHostToDevice);
	free(h_final_colors);
	return d_final_colors;
}

typedef struct
{
	Scene* d_scene;
	Sphere* d_spheres;
} 
SceneReference;

static SceneReference allocate_scene_gpu(Scene* scene)
{
	SceneReference ref;
	int spheres_size = sizeof(Sphere) * scene->sphere_amount;
	hipMalloc(&ref.d_scene, sizeof(Scene));
	hipMalloc(&ref.d_spheres, spheres_size);
	Sphere* h_spheres = scene->spheres;
	scene->spheres = ref.d_spheres;
	hipMemcpy(ref.d_scene, scene, sizeof(Scene), hipMemcpyHostToDevice);
	scene->spheres = h_spheres;
	hipMemcpy(ref.d_spheres, scene->spheres, spheres_size, hipMemcpyHostToDevice);
	return ref;
}

static void free_scene_gpu(SceneReference ref)
{
	hipFree(ref.d_spheres);
	hipFree(ref.d_scene);
}

void render_scene(Bitmap* bitmap, int samples)
{
	struct timespec tstart = {0, 0};
	struct timespec tend = {0, 0};
	clock_gettime(CLOCK_MONOTONIC, &tstart);

	Material white = material_diffuse(vector3_create(1, 1, 1));
	Material white_light = material_emissive(vector3_create(1, 1, 1));
	Material blue = material_diffuse(vector3_create(0, 0, 1));
	Material red = material_diffuse(vector3_create(1, 0, 0));

	Scene* scene = scene_new(4);
	scene->spheres[0] = sphere_create(10, vector3_create(0, -11, 8), white);
	scene->spheres[1] = sphere_create(1, vector3_create(0, 0, 8), white);
	scene->spheres[2] = sphere_create(0.5, vector3_create(-2, -0.75, 7), red);
	scene->spheres[3] = sphere_create(0.5, vector3_create(2, -0.75, 7), blue);
	// scene->spheres[4] = sphere_create(0.75, vector3_create(0, 4, 8), white_light);

	hipDeviceSetLimit(hipLimitMallocHeapSize, 256 * 1024 * 1024);
	int pixels_amount = bitmap->width * bitmap->height;
	int threads_per_block = 256;
	int blocks_amount = (pixels_amount + threads_per_block - 1) / threads_per_block;

	hiprandState* d_states;
	hipMalloc(&d_states, sizeof(hiprandState) * threads_per_block * blocks_amount);
	init_curand_states<<<blocks_amount, threads_per_block>>>(d_states, pixels_amount);

	RenderInfo* d_info = allocate_render_info_gpu(bitmap->width, bitmap->height, 70, 1);

	Vector3* d_final_colors = allocate_final_colors_gpu(pixels_amount);

	Vector3* d_ray_colors;
	hipMalloc(&d_ray_colors, pixels_amount * sizeof(Vector3));

	int* d_ray_statuses;
	hipMalloc(&d_ray_statuses, pixels_amount * sizeof(int));

	Ray* d_rays;
	hipMalloc(&d_rays, sizeof(Ray) * pixels_amount);

	SceneReference ref = allocate_scene_gpu(scene);

	struct timespec tstart_render = {0, 0};
	struct timespec tend_render = {0, 0};
	clock_gettime(CLOCK_MONOTONIC, &tstart_render);

	for (int i = 0; i < samples; i++)
	{
		init_rays<<<blocks_amount, threads_per_block>>>
			(d_rays, d_ray_statuses, d_ray_colors, d_info, d_states, pixels_amount);

		for (int j = 0; j < 5; j++)
		{
			pathtrace_kernel<<<blocks_amount, threads_per_block>>>
				(d_final_colors, d_rays, d_ray_statuses, d_ray_colors, 
				 ref.d_scene, d_states, pixels_amount);		
		}
	}

	clock_gettime(CLOCK_MONOTONIC, &tend_render);
	printf("Render Time: %f seconds\n", 
		    ((double) tend_render.tv_sec + 1.0e-9 * tend_render.tv_nsec) -
		    ((double) tstart_render.tv_sec + 1.0e-9 * tstart_render.tv_nsec));

	hipFree(d_states);
	hipFree(d_rays);
	hipFree(d_info);
	hipFree(d_ray_statuses);
	hipFree(d_ray_colors);
	free_scene_gpu(ref);
	scene_free(scene);

	Pixel* h_pixels = bitmap->pixels;
	Pixel* d_pixels;
	hipMalloc(&d_pixels, sizeof(Pixel) * pixels_amount);
	hipMemcpy(d_pixels, h_pixels, sizeof(Pixel) * pixels_amount, hipMemcpyHostToDevice);

	set_bitmap<<<blocks_amount, threads_per_block>>>(d_final_colors, d_pixels, (float) samples, pixels_amount);
	hipMemcpy(h_pixels, d_pixels, sizeof(Pixel) * pixels_amount, hipMemcpyDeviceToHost);

	hipFree(d_final_colors);
	hipFree(d_pixels);

	clock_gettime(CLOCK_MONOTONIC, &tend);
	printf("Total Time: %f seconds\n", 
		    ((double) tend.tv_sec + 1.0e-9 * tend.tv_nsec) -
		    ((double) tstart.tv_sec + 1.0e-9 * tstart.tv_nsec));
}